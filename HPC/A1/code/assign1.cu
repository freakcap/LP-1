

#include <hip/hip_runtime.h>
#include<iostream>
#include<math.h>

#define n 8

using namespace std;

__global__ void minimum(int *input) {
    int tid = threadIdx.x;
    int step_size = 1;
    int number_of_threads = blockDim.x;
    
    printf("No of threads = %d", number_of_threads);
    
    while(number_of_threads>0) {
        if(tid < number_of_threads) {
            int first = tid*step_size*2;
            int second = first + step_size;
            if(input[second] < input[first])
              input[first] = input[second];
        }
        step_size <<= 1;
        number_of_threads >>= 1;
    }
}

__global__ void maximum(int *input) {
    int tid = threadIdx.x;
    int step_size = 1;
    int number_of_threads = blockDim.x;
    
    while(number_of_threads>0) {
        if(tid < number_of_threads) {
            int first = tid*step_size*2;
            int second = first + step_size;
            if(input[second] > input[first])
              input[first] = input[second];
        }
        step_size <<= 1;
        number_of_threads >>= 1;
    }
}

__global__ void sum(int *input) {
    const int tid = threadIdx.x;
    int step_size = 1;
    int number_of_threads = blockDim.x;
    
    while(number_of_threads > 0) {
        if(tid < number_of_threads) {
            int first = tid * step_size * 2;
            int second = first + step_size;
            
            input[first] += input[second];
        }
        step_size <<= 1;
        number_of_threads >>= 1;
       
    }
}

__global__ void mean_diff_sq(float *input, float mean) {
    input[threadIdx.x] -= mean;
    input[threadIdx.x] *= input[threadIdx.x];
}

__global__ void sum_floats(float *input) {
    int tid = threadIdx.x;
    int step_size = 1;
    int number_of_threads = blockDim.x;
    
    while(number_of_threads > 0) {
        if(tid < number_of_threads) {
            int first = tid * step_size * 2;
            int second = first + step_size;
            
            input[first] += input[second];
        }
        step_size <<= 1;
        number_of_threads >>= 1;
       
    }
}

void copy_int_to_float(float *dest, int *src, int size){
    for(int i=0; i<size; i++)
        dest[i] = float(src[i]);
}

void random_ints(int *input, int size) {
    for(int i=0; i<size; i++)  {
        input[i] = rand()%100;
        cout<<input[i]<<"  ";   
    }
    cout<<endl;

}

int main() {
    //int n=8;
    int size = n*sizeof(int); //calculate no. of bytes for array
        
    int *arr;
    int *arr_d, result;
   
    arr = (int *)malloc(size);
    random_ints(arr, n);
    
    hipMalloc((void **)&arr_d, size);
    
    //MIN
    hipMemcpy(arr_d, arr, size, hipMemcpyHostToDevice);
    
    minimum<<<1,n/2>>>(arr_d);
    
    hipMemcpy(&result, arr_d, sizeof(int), hipMemcpyDeviceToHost);
    
    cout<<"The minimum element is "<<result<<endl;
      
       
    //MAX
    hipMemcpy(arr_d, arr, size, hipMemcpyHostToDevice);
    
    maximum<<<1,n/2>>>(arr_d);
    
    hipMemcpy(&result, arr_d, sizeof(int), hipMemcpyDeviceToHost);
    
    cout<<"The maximum element is "<<result<<endl;
    
    //SUM
    hipMemcpy(arr_d, arr, size, hipMemcpyHostToDevice);
    
    sum<<<1,n/2>>>(arr_d);
    
    hipMemcpy(&result, arr_d, sizeof(int), hipMemcpyDeviceToHost);
    
    cout<<"The sum is "<<result<<endl;
    
    //AVERAGE
    
    float mean = float(result)/n;
    cout<<"The mean is "<<mean<<endl;
    
    //STANDARD DEVIATION
    float *arr_float;
    float *arr_std, stdValue;
    
    arr_float = (float *)malloc(n*sizeof(float));
    hipMalloc((void **)&arr_std, n*sizeof(float));
    
    copy_int_to_float(arr_float, arr, n);
    
    hipMemcpy(arr_std, arr_float, n*sizeof(float), hipMemcpyHostToDevice);
    
    mean_diff_sq <<<1,n>>>(arr_std, mean);
    sum_floats<<<1,n/2>>>(arr_std);
    
    hipMemcpy(&stdValue, arr_std, sizeof(float), hipMemcpyDeviceToHost);
    
    
    stdValue = stdValue / n;
    cout<<"The variance is "<<stdValue<<endl;
    stdValue = sqrt(stdValue);
    
    cout<<"The standard deviation is "<<stdValue<<endl;
    
    hipFree(arr_d);
           
    return 0;
}
