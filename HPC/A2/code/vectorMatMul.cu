
#include <hip/hip_runtime.h>
#include<iostream>
#include<cstdlib>
#include<cmath>
#include<time.h>
using namespace std;


__global__ void matrixVectorMultiplication(int *a, int *b, int *c, int n)
{
    int row=threadIdx.x+blockDim.x*blockIdx.x;
    int sum=0;
   
    if(row<n){
        for(int j=0;j<n;j++)
        {
            sum=sum+a[(j*n)+row]*b[j];
        }
    c[row]=sum;
    }
}
int main()
{
    int *a,*b,*c;
    int *a_dev,*b_dev,*c_dev;
    int n=10;
    
    a=new int[n*n];
    b=new int[n];
    c=new int[n];
    int *d=new int[n];
    int size=n*sizeof(int);
    hipMalloc(&a_dev,size*size);
    hipMalloc(&b_dev,size);
    hipMalloc(&c_dev,size);
    

    cout<<"\n\nMatrix is :\n\n";
    for(int i=0;i<n;i++)
    {
        for(int j=0;j<n;j++)
        {
            a[i*n+j]= i*n+j+1; //rand()%n;
            cout<<a[i*n+j]<<" ";
        }
        
        b[i]= i+1; //rand()%n;
        cout<<"\n";
       // d[i]=a[i]+b[i];
    }
    
    cout<<"\n\nVector is: \n\n";
    for(int i=0;i<n;i++)
        cout<<b[i]<<" ";
    cout<<"\n\n";
    hipMemcpy(a_dev,a,size*size,hipMemcpyHostToDevice);
    hipMemcpy(b_dev,b,size,hipMemcpyHostToDevice);
    
    
    dim3 threadsPerBlock(n, n);
    dim3 blocksPerGrid(1, 1);
    
    if(n*n>512){
        threadsPerBlock.x=512;
        threadsPerBlock.y=512;
        blocksPerGrid.x=ceil((double)n/(double)threadsPerBlock.x);
        blocksPerGrid.y=ceil((double)n/(double)threadsPerBlock.y);
    }
    
    matrixVectorMultiplication<<<n/256 +1,256>>>(a_dev,b_dev,c_dev,n);
    
    hipMemcpy(c,c_dev,size,hipMemcpyDeviceToHost);
    
    //CPU matrixVector multiplication
    clock_t t=clock();
    int sum=0;
    for(int row=0;row<n;row++)
    {
        sum=0;
        for(int col=0;col<n;col++)
        {
              sum=sum+a[col*n+row]*b[col];  
            
        }
      d[row]=sum;
    }
    t=clock()-t;
        cout<<"\nCPU Time Elapsed:  "<<((double)t);      //((double)t)/CLOCKS_PER_SEC;

    
    int error=0;
    cout<<"\n\n";
    for(int i=0;i<n;i++){
        error+=d[i]-c[i];
       cout<<" gpu "<<c[i]<<" CPU "<<d[i]<<endl;
    }
    
    cout<<"\nError : "<<error<<"\n\n";
    
    
    return 0;
}

/*
Output
==11960== NVPROF is profiling process 11960, command: ./a.out


Matrix is :

1 2 3 4 5 6 7 8 9 10 
11 12 13 14 15 16 17 18 19 20 
21 22 23 24 25 26 27 28 29 30 
31 32 33 34 35 36 37 38 39 40 
41 42 43 44 45 46 47 48 49 50 
51 52 53 54 55 56 57 58 59 60 
61 62 63 64 65 66 67 68 69 70 
71 72 73 74 75 76 77 78 79 80 
81 82 83 84 85 86 87 88 89 90 
91 92 93 94 95 96 97 98 99 100 


Vector is: 

1 2 3 4 5 6 7 8 9 10 


CPU Time Elapsed:  3

 gpu 3355 CPU 3355
 gpu 3410 CPU 3410
 gpu 3465 CPU 3465
 gpu 3520 CPU 3520
 gpu 3575 CPU 3575
 gpu 3630 CPU 3630
 gpu 3685 CPU 3685
 gpu 3740 CPU 3740
 gpu 3795 CPU 3795
 gpu 3850 CPU 3850

Error : 0

==11960== Profiling application: ./a.out
==11960== Profiling result:
            Type  Time(%)      Time     Calls       Avg       Min       Max  Name
 GPU activities:   48.69%  4.1910us         1  4.1910us  4.1910us  4.1910us  matrixVectorMultiplication(int*, int*, int*, int)
                   30.86%  2.6560us         2  1.3280us  1.0880us  1.5680us  [CUDA memcpy HtoD]
                   20.45%  1.7600us         1  1.7600us  1.7600us  1.7600us  [CUDA memcpy DtoH]
      API calls:   99.71%  200.37ms         3  66.791ms  5.1320us  200.36ms  cudaMalloc
                    0.16%  313.26us        97  3.2290us     124ns  206.98us  cuDeviceGetAttribute
                    0.05%  104.47us         1  104.47us  104.47us  104.47us  cuDeviceTotalMem
                    0.03%  64.413us         1  64.413us  64.413us  64.413us  cuDeviceGetName
                    0.03%  62.632us         3  20.877us  19.062us  22.537us  cudaMemcpy
                    0.02%  32.072us         1  32.072us  32.072us  32.072us  cudaLaunchKernel
                    0.00%  4.1760us         1  4.1760us  4.1760us  4.1760us  cuDeviceGetPCIBusId
                    0.00%  2.1090us         3     703ns     121ns  1.7920us  cuDeviceGetCount
                    0.00%     988ns         2     494ns     137ns     851ns  cuDeviceGet
                    0.00%     210ns         1     210ns     210ns     210ns  cuDeviceGetUuid

*/